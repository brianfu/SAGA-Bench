#include "builder.h"

#include <iostream>
#include <fstream>

#include "topAlg_cu.h"
#include "topDataStruc_cu.h"
#include "../common/timer.h"

void* dequeAndInsertEdge(
		std::string dtype,
		dataStruc *ds,
    EdgeBatchQueue *q,
    std::mutex *q_lock,
    std::string algorithm,
    bool *still_reading)
{	
	//std::cout << "Thread dequeAndInsertEdge: on CPU " << sched_getcpu() << "\n";
    Algorithm alg(algorithm, ds, dtype);
    int batch = 0;
    EdgeList el;
    q_lock->lock();
    while (*still_reading || !q->empty()) {		
	if (!q->empty()) {		
	    el = q->front();
	    q->pop();
	    q_lock->unlock();
		Timer t;
		t.Start();
		ds->update(el);	
		
		t.Stop();    
        ofstream out("Update.csv", std::ios_base::app);   
        out << t.Seconds() << std::endl;    
        out.close();	
	    std::cout << "Updated Batch: " << batch << std::endl;
	    batch++;
	    alg.performAlg();

		if(ds->num_edges == 234370166)
		{
			ofstream myfile;
			myfile.open("/home/tmathew/sfuhome/dataset/cudaBfsFromScratch" + std::to_string(batch) + ".csv");
			for (int i=0; i < ds->property.size(); i++)
			{
				myfile << i << ", " << ds->property[i] << "\n";
				
			}
			myfile.close();
		}
	} else {		
	    q_lock->unlock();		
	    std::this_thread::sleep_for(std::chrono::milliseconds(1));		
	}
	q_lock->lock();
    }
    q_lock->unlock();
    
    // ##################### CORRECTNESS CHECK ############################
    // LJ: batch == 138
    // Orkut: batch == 235
    // Pokec: batch == 62
    // Wiki: batch == 58
    // 15_30m: batch == 61

    /*for (int64_t i = 0; i < ds->num_nodes; ++i) {
	std::cout << "Property[" << i << "] = "
		  << ds->property[i] << std::endl;
    }*/

    /*if ((algorithm == "prdyn") && (dtype == "adListChunked")) {
	ofstream out("PRDynAdListOrkut.csv"); 
	for(int64_t i =0; i < ds->num_nodes; i++){
	    out << ds->property[i] << endl;
	}                       
	out.close(); 
    } else if ((algorithm == "prfromscratch") && (dtype == "adListChunked")) {
	ofstream out("PRStatAdListOrkut.csv"); 
	for(int64_t i =0; i < ds->num_nodes; i++){
	    out << ds->property[i] << endl;
	}                       
	out.close();
    } else if ((algorithm == "prdyn") && (dtype == "degAwareRHH")) {
	ofstream out("PRDynDarhhOrkut.csv"); 
	for(int64_t i =0; i < ds->num_nodes; i++){
	    out << ds->property[i] << endl;
	}                       
	out.close();
    } else if ((algorithm == "prfromscratch") && (dtype == "degAwareRHH")) {
	ofstream out("PRStatDarhhOrkut.csv");
	for(int64_t i =0; i < ds->num_nodes; i++){
	    out << ds->property[i] << endl;
	}                       
	out.close();
    }*/

    return 0;
}
