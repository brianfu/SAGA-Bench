#ifndef TOPDATASTRUC_H_
#define TOPDATASTRUC_H_

#include <string>

#include "adListCuda_.cu"
// #include "adListShared.h"
#include "stinger.h"
#include "darhh.h"
#include "adListChunked.h"

dataStruc* createDataStruc(const std::string& type, bool weighted, bool directed, int64_t num_nodes, int64_t num_threads);
#endif