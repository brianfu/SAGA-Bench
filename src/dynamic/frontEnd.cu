#include <unistd.h>
#include <fstream>
#include <cstring>
#include <mutex>
#include <thread>

#include "builder.h"
#include "fileReader.h"
#include "topDataStruc_cu.h"
#include "parser.h"

/* Main thread that launches everything else */

int main(int argc, char* argv[])
{    
    cmd_args opts = parse(argc, argv);
    ifstream file(opts.filename);
    if (!file.is_open()) {
        cout << "Couldn't open file " << opts.filename << endl;
	exit(-1);
    }    

    std::mutex q_lock;
    
    EdgeBatchQueue queue;
    bool loop = true;  
    dataStruc* struc = createDataStruc(opts.type, opts.weighted, opts.directed, opts.num_nodes, opts.num_threads);    
    std::thread t1(dequeAndInsertEdge, opts.type, struc, &queue, &q_lock, opts.algorithm, &loop);   
    
    cpu_set_t cpuset;
    CPU_ZERO(&cpuset);
    CPU_SET(1, &cpuset);

    int rc = pthread_setaffinity_np(t1.native_handle(), sizeof(cpu_set_t), &cpuset);
    
    if (rc != 0) {
        std::cerr << "Error calling pthread_setaffinity_np: " << rc << "\n";
    }

    int batch_id = 0;
    NodeID lastAssignedNodeID = -1;
    MapTable VMAP;

    while (!file.eof()) {        
        EdgeList el = readBatchFromCSV(
	    file,
	    opts.batch_size,
	    batch_id,
	    opts.weighted,
	    VMAP,
	    lastAssignedNodeID);
	q_lock.lock();     
        queue.push(el);
	q_lock.unlock();
	batch_id++;          
    }
    file.close();

    bool allEmpty = false;
    while (!allEmpty) {   
        q_lock.lock();
	allEmpty = queue.empty();
	q_lock.unlock();
	sleep(20);
    }
    
    loop = false;
    t1.join();
    
    //cout << "Started printing queues " << endl;
    //printEdgeBatchQueue(queue);
    //cout << "Done printing queues " << endl;    
    struc->print();
}